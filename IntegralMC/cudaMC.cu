#include "hip/hip_runtime.h"
#include "stdafx.h"


using namespace std;

cudaMC::cudaMC()
{
}
int cudaMC::CUDASize() {
	int CUDASize;
	hipGetDeviceCount(&CUDASize);
	return CUDASize;
}
int cudaMC::CUDAName_length() {
	return 256;
}



//[TEST]///////////////////////////////////

__global__
void my_kernel() {
	clock_t ct1, ct2, ctend;

	ct1 = clock();

	while (ctend < 1000) {
		ct2 = clock();
		ctend = ct2 - ct1;
	}

}

void cudaMC::call_kernel() {
	// prepare data for the kernel, e.g. allocating memory, copying from host to device, etc.
	//czas cuda

	// run kernel
	my_kernel << < 10,10>> >();

	// copy results from device to host, clean up, etc.
}

void cudaMC::cuRand_test() {

	hiprandGenerator_t genGPU;
	hiprandGenerator_t genCPU;

	hiprandCreateGenerator(&genGPU, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(genGPU, 1234ULL);
	hiprandCreateGeneratorHost(&genCPU, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(genCPU, 1234ULL);

	const int n = 10;
	double CPU[n];
	double GPU[n];

	double* d_GPU;
	hipMalloc(&d_GPU, n * sizeof(double));

	hiprandGenerateUniformDouble(genCPU, CPU, n);
	hiprandGenerateUniformDouble(genGPU, d_GPU, n);
	hipMemcpy(GPU, d_GPU, n * sizeof(double), hipMemcpyDeviceToHost);

	for (int i = 0; i < n; ++i) {
		cout << CPU[i] << ' ' << GPU[i] << endl;
	}

	hiprandDestroyGenerator(genGPU);
	hiprandDestroyGenerator(genCPU);
	hipFree(d_GPU);
}